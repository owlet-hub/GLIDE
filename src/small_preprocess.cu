#include "load.cuh"
#include "params.cuh"
#include "partition.cuh"
#include <fstream>

/**
 * @brief Main function for data preprocessing pipeline for small-scale data
 *
 * @param argc Number of command line arguments
 * @param argv Command line arguments:
 *             [0] Program name
 *             [1] Input data file path
 *             [2] Output preprocess file base path
 *             [3] Result file path
 *             [4] Number of centroids (uint32_t)
 *             [5] Boundary factor for boundary point identification (float)
 *             [6] Sample factor for sampling-based cluster (float)
 *             [7] Distance metric ("Euclidean" or "Cosine")
 *
 * @return int Program exit status (0 for success, non-zero for failure)
 */
int main(int argc, char **argv) {
    if (argc != 8) {
        std::cout << argv[0]
                  << "data_file preprocess_file result_file centroid_num boundary_factor sample_factor metric"
                  << std::endl;
        exit(-1);
    }

    hipSetDevice(1);
    raft::device_resources handle;
    auto stream_pool = std::make_shared<rmm::cuda_stream_pool>(32);
    raft::resource::set_cuda_stream_pool(handle, stream_pool);

    std::string data_file(argv[1]);
    std::string preprocess_file(argv[2]);
    std::string result_file(argv[3]);
    std::string centroid_file = preprocess_file + ".centroid";
    std::string segment_file = preprocess_file + ".segment";
    std::string map_file = preprocess_file + ".map";
    std::string reorder_file = preprocess_file + ".reorder";

    PartitionParameter partition_param;
    partition_param.centroid_num = std::stoi(argv[4]);
    partition_param.boundary_factor = std::stof(argv[5]);
    partition_param.sample_factor = std::stof(argv[6]);
    std::string metric(argv[7]);
    if (metric == "Euclidean") {
        partition_param.metric = Metric::Euclidean;
    } else if (metric == "Cosine") {
        partition_param.metric = Metric::Cosine;
    }

    auto h_data = load_matrix_data<float, uint32_t>(data_file);
    uint32_t num = h_data.extent(0);
    uint32_t dim = h_data.extent(1);

    std::optional<raft::device_matrix<float>> d_reorder_data;
    std::optional<raft::device_vector<uint32_t>> d_map;
    auto h_segment_start = raft::make_host_vector<uint32_t>(partition_param.centroid_num + 1);
    auto h_segment_length = raft::make_host_vector<uint32_t>(partition_param.centroid_num + 1);
    auto d_centroids = raft::make_device_matrix<float, int>(handle, partition_param.centroid_num, dim);


    auto d_data = raft::make_device_matrix<float>(handle, num, dim);
    raft::copy(d_data.data_handle(), h_data.data_handle(),
               num * dim, raft::resource::get_cuda_stream(handle));

    preprocess<float, uint32_t>(handle, partition_param, d_data.view(), d_reorder_data, d_map, h_segment_start.view(),
                                h_segment_length.view(), d_centroids.view(), result_file);

    auto h_centroids = raft::make_host_matrix<float>(partition_param.centroid_num, dim);
    raft::copy(h_centroids.data_handle(), d_centroids.data_handle(),
               h_centroids.size(), raft::resource::get_stream_from_stream_pool(handle));
    auto h_map = raft::make_host_vector<uint32_t>(d_map->size());
    raft::copy(h_map.data_handle(), d_map->data_handle(),
               h_map.size(), raft::resource::get_stream_from_stream_pool(handle));
    auto h_reorder_data = raft::make_host_matrix<float>(d_reorder_data->extent(0), d_reorder_data->extent(1));
    raft::copy(h_reorder_data.data_handle(), d_reorder_data->data_handle(),
               h_reorder_data.size(), raft::resource::get_stream_from_stream_pool(handle));

    save_matrix_data<float, uint32_t>(centroid_file, h_centroids.view());
    save_segment(segment_file, h_segment_start.view(), h_segment_length.view());
    save_vector_data(map_file, h_map.view());
    save_matrix_data<float, uint32_t>(reorder_file, h_reorder_data.view());
}