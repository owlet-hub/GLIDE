#include "load.cuh"
#include "glide_large_impl.cuh"
#include <fstream>

uint32_t calculate_recall(raft::host_matrix_view<uint32_t> neighbors,
                          raft::host_matrix_view<uint32_t> truth,
                          uint32_t top_k) {
    uint32_t query_num = neighbors.extent(0);
    float total_recall = 0.0;

    for (uint32_t query_id = 0; query_id < query_num; query_id++) {
        uint32_t correct_count = 0;
        for (uint32_t k = 0; k < top_k; k++) {
            uint32_t neighbor_id = neighbors(query_id, k);
            for (uint32_t t = 0; t < top_k; t++) {
                if (truth(query_id, t) == neighbor_id) {
                    correct_count++;
                    break;
                }
            }
        }
        float recall = static_cast<float>(correct_count) / top_k;
        total_recall += recall;
    }
    std::cout << "total_recall: " << total_recall / query_num << std::endl;
    return total_recall / query_num;
}

int main(int argc, char **argv) {
    if (argc != 11) {
        std::cout << argv[0]
                  << "preprocess_file query_file truth_file graph_base_file result_file metric topk search_beam min_segment_num boundary_factor"
                  << std::endl;
        exit(-1);
    }

    hipSetDevice(1);
    raft::device_resources handle;
    auto stream_pool = std::make_shared<rmm::cuda_stream_pool>(32);
    raft::resource::set_cuda_stream_pool(handle, stream_pool);

    std::string preprocess_file(argv[1]);
    std::string segment_file = preprocess_file + ".segment";
    std::string reorder_file = preprocess_file + ".reorder";
    std::string map_file = preprocess_file + ".map";
    std::string centroid_file = preprocess_file + ".centroids";
    std::string query_file(argv[2]);
    std::string truth_file(argv[3]);
    std::string graph_base_file(argv[4]);
    std::string result_file(argv[5]);
    std::string graph_file = graph_base_file + ".graph";
    std::string start_point_file = graph_base_file + ".sp";
    std::string metric(argv[6]);
    Metric index_metric;
    if (metric == "Euclidean") {
        index_metric = Metric::Euclidean;
    } else if (metric == "Cosine") {
        index_metric = Metric::Cosine;
    }

    SearchParameter search_param;
    search_param.topk = std::stoi(argv[7]);
    search_param.beam = std::stoi(argv[8]);
    uint32_t min_segment_num = std::stoi(argv[9]);
    float boundary_factor = std::stof(argv[10]);

    std::ofstream result_out(result_file, std::ios::app);
    result_out << search_param.beam << ",";
    result_out.close();

    GLIDE_large index(handle, index_metric,
                      reorder_file, map_file, centroid_file,
                      segment_file, start_point_file, graph_file);

    adjust_search_params(search_param.min_iterations, search_param.max_iterations, search_param.beam);
    search_param.hash_bit = calculate_hash_bitlen(search_param.beam, index.graph_degree(),
                                                  search_param.hash_max_fill_rate,
                                                  search_param.hashmap_min_bitlen);
    search_param.hash_reset_interval = calculate_hash_reset_interval(search_param.beam,
                                                                     index.graph_degree(),
                                                                     search_param.hash_max_fill_rate,
                                                                     search_param.hash_bit);

    auto query = load_data<uint8_t, uint32_t>(query_file);
    auto d_query = raft::make_device_matrix<uint8_t, uint32_t>(handle, query.extent(0), query.extent(1));
    raft::copy(d_query.data_handle(), query.data_handle(),
               query.size(), raft::resource::get_stream_from_stream_pool(handle));
    auto result_ids = raft::make_host_matrix<uint32_t, uint32_t>(d_query.extent(0), search_param.topk);
    auto result_distances = raft::make_host_matrix<float, uint32_t>(d_query.extent(0), search_param.topk);

    index.search(search_param, min_segment_num, boundary_factor, d_query.view(), result_ids.view(),
                 result_distances.view(), result_file);

    auto truth = load_data<uint32_t, uint32_t>(truth_file);
    float recall = calculate_recall(result_ids.view(), truth.view(), search_param.topk);

    std::ofstream result;
    result.open(result_file, std::ios::app);
    result << recall << std::endl;
    result.close();
}